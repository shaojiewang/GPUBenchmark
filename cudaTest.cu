#include "hip/hip_runtime.h"
#include "cudaTest.h"
#include <array>
#include <math.h>
#include <chrono>
#include <iostream>

using namespace std;
using namespace std::chrono;

using testType = float;

#define nDim 1 

#define BLOCKSIZE_T 128

template<int BlockSize, typename T, class Tensor>
__global__ void TensorDataTransfer(Tensor A, Tensor B){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    B.t_data[tid] = A.t_data[tid];
}


void cudaBWTest()
{
    std::array<int, nDim> lengths;
    std::array<int, nDim> strides;
    for(int i = 0; i < nDim; i++){
        lengths[i] = 1024;
        if(i == 0)
            strides[i] = 1;
        else
            strides[i] = lengths[i - 1];
    }
    testTensor<testType, nDim> HostA(lengths, strides);
    testTensor<testType, nDim> HostB(lengths, strides);

    HostA.t_data = static_cast<testType* >(malloc(HostA.get_tensor_space_size() * sizeof(testType)));
    HostB.t_data = static_cast<testType* >(malloc(HostB.get_tensor_space_size() * sizeof(testType)));


    testTensor<testType, nDim> A_device;
    for(int i = 0; i < nDim; i++){
        A_device.lengths[i] = HostA.lengths[i];
        A_device.strides[i] = HostA.strides[i];
    }
    hipMalloc(&A_device.t_data, A_device.get_tensor_space_size() * sizeof(testType));
    hipMemcpy(A_device.t_data, HostA.t_data, A_device.get_tensor_space_size() * sizeof(testType), hipMemcpyHostToDevice);


    testTensor<testType, nDim> B_device;
    for(int i = 0; i < nDim; i++){
        B_device.lengths[i] = HostB.lengths[i];
        B_device.strides[i] = HostB.strides[i];
    }
    hipMalloc(&B_device.t_data, B_device.get_tensor_space_size() * sizeof(testType));
    //hipMemcpy(B_device.t_data, B.t_data, B_device.get_tensor_space_size() * sizeof(testType), hipMemcpyHostToDevice);

    size_t ele_size = HostA.get_tensor_size();

    dim3 dim_block(BLOCKSIZE_T, 1, 1);
    dim3 dim_grid(ele_size / BLOCKSIZE_T, 1, 1);

    auto startTime = high_resolution_clock::now();

    TensorDataTransfer<BLOCKSIZE_T, testType, decltype(B_device)><<<dim_grid, dim_block>>>(A_device, B_device);

    auto endTime = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(endTime - startTime); 
    std::cout << "cuda runtime" <<double(duration.count()) / 1000 << "ms" << std::endl;

    hipMemcpy(HostB.t_data, B_device.t_data, B_device.get_tensor_space_size() * sizeof(testType), hipMemcpyDeviceToHost);
    hipFree(A_device.t_data);
    hipFree(B_device.t_data);

    free(HostA.t_data);
    free(HostB.t_data);

}